#include "hip/hip_runtime.h"
// CUDA C/C++ implementation for Accelerating Graph Betweenness Centrality for Sparse Graphs

#include <iostream>
#include <hip/hip_runtime.h>
#include "Graph.h"

#define MAX_THREAD_COUNT 1024
#define CEIL(a, b) ((a - 1) / b + 1)
// Max device memory : 4 GB
#define MAX_MEMORY ((long long)4e9)

using namespace std;

#define catchCudaError(error) { gpuAssert((error), __FILE__, __LINE__); }

float device_time_taken;

// Catch Cuda errors
inline void gpuAssert(hipError_t error, const char *file, int line,  bool abort = false)
{
    if (error != hipSuccess)
    {
        printf("\n====== Cuda Error Code %i ======\n %s in CUDA %s\n", error, hipGetErrorString(error));
        printf("\nIn file :%s\nOn line: %d", file, line);
        
        if(abort)
            exit(-1);
    }
}

__global__ void betweennessCentralityKernel(Graph *graph, float *bwCentrality, int nodeCount,
            int *sigma, int *distance, float *dependency, int *Q, int *Qpointers) {
    
    int idx = threadIdx.x;
    if(idx >= nodeCount)
        return;
    
    __shared__ int s;
    __shared__ int Q_len;
    __shared__ int Qpointers_len;
    __shared__ int noOfBlocks;

    if(idx == 0) {
        s = blockIdx.x - gridDim.x;
        noOfBlocks = gridDim.x;
        // printf("Progress... %3d%%", 0);
    }
    __syncthreads();
    
    while(s < nodeCount - noOfBlocks)
    {
        if(idx == 0)
        {
            s += noOfBlocks;
            // printf("\rProgress... %5.2f%%", (s+1)*100.0/nodeCount);
            // printf("Node %d\n", s);
            
            Q[0 + (blockIdx.x * nodeCount)] = s;
            Q_len = 1;
            Qpointers[0 + (blockIdx.x * nodeCount)] = 0;
            Qpointers[1 + (blockIdx.x * nodeCount)] = 1;
            Qpointers_len = 1;
        }
        __syncthreads();

        for(int v=idx; v<nodeCount; v+=blockDim.x)
        {
            if(v == s)
            {
                distance[v + (blockIdx.x * nodeCount)] = 0;
                sigma[v + (blockIdx.x * nodeCount)] = 1;
            }
            else
            {
                distance[v + (blockIdx.x * nodeCount)] = INT_MAX;
                sigma[v + (blockIdx.x * nodeCount)] = 0;
            }
            dependency[v + (blockIdx.x * nodeCount)] = 0.0;
        }
        __syncthreads();
        
        // BFS
        while(true)
        {
            __syncthreads();
            for(int k=idx; k<Qpointers[Qpointers_len + (blockIdx.x * nodeCount)]; k+=blockDim.x) 
            {
                if(k < Qpointers[Qpointers_len -1 + (blockIdx.x * nodeCount)])
                    continue;

                int v = Q[k + (blockIdx.x * nodeCount)];
                for(int r = graph->adjacencyListPointers[v]; r < graph->adjacencyListPointers[v + 1]; r++)
                {
                    int w = graph->adjacencyList[r];
                    if(atomicCAS(&distance[w + (blockIdx.x * nodeCount)], INT_MAX, distance[v + (blockIdx.x * nodeCount)] +1) == INT_MAX)
                    {
                        int t = atomicAdd(&Q_len, 1);
                        Q[t + (blockIdx.x * nodeCount)] = w;
                    }
                    if(distance[w + (blockIdx.x * nodeCount)] == (distance[v + (blockIdx.x * nodeCount)]+1))
                    {
                        atomicAdd(&sigma[w + (blockIdx.x * nodeCount)], sigma[v + (blockIdx.x * nodeCount)]);
                    }
                }
            }
            __syncthreads();

            if(Q_len == Qpointers[Qpointers_len + (blockIdx.x * nodeCount)])
                break;

            if(idx == 0)
            {
                Qpointers_len++;
                Qpointers[Qpointers_len + (blockIdx.x * nodeCount)] = Q_len;
            }
            __syncthreads();
        }
        __syncthreads();
        
        // Reverse BFS
        while(Qpointers_len > 0)
        {
            for(int k=idx; k < Qpointers[Qpointers_len + (blockIdx.x * nodeCount)]; k+=blockDim.x) 
            {
                if(k < Qpointers[Qpointers_len -1 + (blockIdx.x * nodeCount)])
                    continue;

                int v = Q[k + (blockIdx.x * nodeCount)];
                for(int r = graph->adjacencyListPointers[v]; r < graph->adjacencyListPointers[v + 1]; r++)
                {
                    int w = graph->adjacencyList[r];
                    if(distance[w + (blockIdx.x * nodeCount)] == (distance[v + (blockIdx.x * nodeCount)] + 1))
                    {
                        if (sigma[w + (blockIdx.x * nodeCount)] != 0)
                            dependency[v + (blockIdx.x * nodeCount)] += (sigma[v + (blockIdx.x * nodeCount)] * 1.0 / sigma[w + (blockIdx.x * nodeCount)]) * (1 + dependency[w + (blockIdx.x * nodeCount)]);
                    }
                }
                if (v != s)
                {
                    // Each shortest path is counted twice. So, each partial shortest path dependency is halved.
                    atomicAdd(bwCentrality + v, dependency[v + (blockIdx.x * nodeCount)] / 2);
                }
            }
            __syncthreads();

            if(idx == 0)
                Qpointers_len--;

            __syncthreads();
        }
    }
}

float *betweennessCentrality(Graph *graph, int nodeCount)
{
    float *bwCentrality = new float[nodeCount]();
    float *device_bwCentrality, *dependency;
    int *sigma, *distance, *Q, *Qpointers;

    const int BLOCK_COUNT = MAX_MEMORY / (4 * 5 * nodeCount);
    // pritnf(">> %d\n", BLOCK_COUNT);

    //TODO: Allocate device memory for bwCentrality
    catchCudaError(hipMalloc((void **)&device_bwCentrality, sizeof(float) * nodeCount));
    catchCudaError(hipMalloc((void **)&sigma, sizeof(int) * nodeCount * BLOCK_COUNT));
    catchCudaError(hipMalloc((void **)&distance, sizeof(int) * nodeCount * BLOCK_COUNT));
    catchCudaError(hipMalloc((void **)&Q, sizeof(int) * (nodeCount) * BLOCK_COUNT));
    catchCudaError(hipMalloc((void **)&Qpointers, sizeof(int) * (nodeCount) * BLOCK_COUNT));
    catchCudaError(hipMalloc((void **)&dependency, sizeof(float) * nodeCount * BLOCK_COUNT));

    catchCudaError(hipMemcpy(device_bwCentrality, bwCentrality, sizeof(float) * nodeCount, hipMemcpyHostToDevice));

    // Timer
    hipEvent_t device_start, device_end;
    catchCudaError(hipEventCreate(&device_start));
    catchCudaError(hipEventCreate(&device_end));
    catchCudaError(hipEventRecord(device_start));
    

    betweennessCentralityKernel<<<BLOCK_COUNT, MAX_THREAD_COUNT>>>(graph, device_bwCentrality, nodeCount, sigma, distance, dependency, Q, Qpointers);
    hipDeviceSynchronize();

    // Timer
    catchCudaError(hipEventRecord(device_end));
    catchCudaError(hipEventSynchronize(device_end));
    hipEventElapsedTime(&device_time_taken, device_start, device_end);

    // Copy back and free memory
    catchCudaError(hipMemcpy(bwCentrality, device_bwCentrality, sizeof(float) * nodeCount, hipMemcpyDeviceToHost));
    catchCudaError(hipFree(device_bwCentrality));
    catchCudaError(hipFree(sigma));
    catchCudaError(hipFree(dependency));
    catchCudaError(hipFree(distance));
    catchCudaError(hipFree(Q));
    catchCudaError(hipFree(Qpointers));
    return bwCentrality;
}

int main(int argc, char *argv[])
{

    if (argc < 2)
    {
        cout << "Please use correct format while execution" << endl;
        return 0;
    }

    // char choice;
    // cout << "Would you like to print the Graph Betweenness Centrality for all nodes? (y/n) ";
    // cin >> choice;

    freopen(argv[1], "r", stdin);

    Graph *host_graph = new Graph();
    Graph *device_graph;

    catchCudaError(hipMalloc((void **)&device_graph, sizeof(Graph)));
    host_graph->readGraph();

    int nodeCount = host_graph->getNodeCount();
    int edgeCount = host_graph->getEdgeCount();
    catchCudaError(hipMemcpy(device_graph, host_graph, sizeof(Graph), hipMemcpyHostToDevice));

    // Copy Adjancency List to device
    int *adjacencyList;
    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&adjacencyList, sizeof(int) * (2 * edgeCount + 1)));
    catchCudaError(hipMemcpy(adjacencyList, host_graph->adjacencyList, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));
    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->adjacencyList), &adjacencyList, sizeof(int *), hipMemcpyHostToDevice));

    // Copy Adjancency List Pointers to device
    int *adjacencyListPointers;
    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&adjacencyListPointers, sizeof(int) * (nodeCount + 1)));
    catchCudaError(hipMemcpy(adjacencyListPointers, host_graph->adjacencyListPointers, sizeof(int) * (nodeCount + 1), hipMemcpyHostToDevice));
    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->adjacencyListPointers), &adjacencyListPointers, sizeof(int *), hipMemcpyHostToDevice));

    float *bwCentrality = betweennessCentrality(device_graph, nodeCount);

    float maxBetweenness = -1;
    for (int i = 0; i < nodeCount; i++)
    {
        maxBetweenness = max(maxBetweenness, bwCentrality[i]);
        // if (choice == 'y' || choice == 'Y')
        //     printf("Node %d => Betweeness Centrality %0.2lf\n", i, bwCentrality[i]);
    }

    printf("Maximum Betweenness Centrality = %0.2lf\n", maxBetweenness);
    printf("Time Taken (Parallel) = %d ms\n", (int)device_time_taken);

    if (argc == 3)
    {
        freopen(argv[2], "w", stdout);
        for (int i = 0; i < nodeCount; i++)
            cout << bwCentrality[i] << " ";
        cout << endl;
    }

    // Free all memory
    catchCudaError(hipFree(adjacencyList));
    catchCudaError(hipFree(adjacencyListPointers));
    catchCudaError(hipFree(device_graph));
}