#include "hip/hip_runtime.h"
// CUDA C/C++ implementation for Accelerating Graph Betweenness Centrality for Sparse Graphs

#include <iostream>
#include <hip/hip_runtime.h>
#include "Graph.h"

#define MAX_THREAD_COUNT 1024
#define CEIL(a, b) ((a - 1) / b + 1)

using namespace std;

#define catchCudaError(error) { gpuAssert((error), __FILE__, __LINE__); }

float device_time_taken;

// Catch Cuda errors
inline void gpuAssert(hipError_t error, const char *file, int line,  bool abort = false)
{
    if (error != hipSuccess)
    {
        printf("\n====== Cuda Error Code %i ======\n %s in CUDA %s\n", error, hipGetErrorString(error));
        printf("\nIn file :%s\nOn line: %d", file, line);
        
        if(abort)
            exit(-1);
    }
}

__global__ void betweennessCentralityKernel(Graph *graph, double *bwCentrality, int nodeCount,
            int *sigma, int *distance, double *dependency) {
    
    int idx = threadIdx.x;
    if(idx >= nodeCount)
        return;
    
    __shared__ int s;
    __shared__ int current_depth;
    __shared__ bool done;

    if(idx == 0) {
        s = -1;
        // printf("Progress... %3d%%", 0);
    }
    __syncthreads();

    while(s < nodeCount -1)
    {    
        if(idx == 0)
        {
            ++s;
            // printf("\rProgress... %5.2f%%", (s+1)*100.0/nodeCount);
            done = false;
            current_depth = -1;
        }
        __syncthreads();

        for(int v=idx; v<nodeCount; v+=blockDim.x)
        {
            if(v == s)
            {
                distance[v] = 0;
                sigma[v] = 1;
            }
            else
            {
                distance[v] = INT_MAX;
                sigma[v] = 0;
            }
            dependency[v] = 0.0;
        }
        __syncthreads();
        
       
        // BFS    
        while(!done)
        {
            if(idx == 0){
                current_depth++;
            }
            done = true;
            __syncthreads();

            for(int v=idx; v<nodeCount; v+=blockDim.x)
            {
                if(distance[v] == current_depth)
                {
                    for(int r = graph->adjacencyListPointers[v]; r < graph->adjacencyListPointers[v + 1]; r++)
                    {
                        int w = graph->adjacencyList[r];
                        if(distance[w] == INT_MAX)
                        {
                            distance[w] = distance[v] + 1;
                            done = false;
                        }
                        if(distance[w] == (distance[v] + 1))
                        {
                            atomicAdd(&sigma[w], sigma[v]);
                        }
                    }
                }
            }
            __syncthreads();
        }

        // Reverse BFS
        while(current_depth)
        {
            if(idx == 0){
                current_depth--;
            }
            __syncthreads();

            for(int v=idx; v<nodeCount; v+=blockDim.x) 
            {
                if(distance[v] == current_depth)
                {
                    for(int r = graph->adjacencyListPointers[v]; r < graph->adjacencyListPointers[v + 1]; r++)
                    {
                        int w = graph->adjacencyList[r];
                        if(distance[w] == (distance[v] + 1))
                        {
                            if (sigma[w] != 0)
                                dependency[v] += (sigma[v] * 1.0 / sigma[w]) * (1 + dependency[w]);
                        }
                    }
                    if (v != s)
                    {
                        // Each shortest path is counted twice. So, each partial shortest path dependency is halved.
                        bwCentrality[v] += dependency[v] / 2;
                    }
                }
            }
            __syncthreads();
        }
    }
}

double *betweennessCentrality(Graph *graph, int nodeCount)
{
    double *bwCentrality = new double[nodeCount]();
    double *device_bwCentrality, *dependency;
    int *sigma, *distance;

    //TODO: Allocate device memory for bwCentrality
    catchCudaError(hipMalloc((void **)&device_bwCentrality, sizeof(double) * nodeCount));
    catchCudaError(hipMalloc((void **)&sigma, sizeof(int) * nodeCount));
    catchCudaError(hipMalloc((void **)&distance, sizeof(int) * nodeCount));
    catchCudaError(hipMalloc((void **)&dependency, sizeof(double) * nodeCount));
    catchCudaError(hipMemcpy(device_bwCentrality, bwCentrality, sizeof(double) * nodeCount, hipMemcpyHostToDevice));

    // Timer
    hipEvent_t device_start, device_end;
    catchCudaError(hipEventCreate(&device_start));
    catchCudaError(hipEventCreate(&device_end));
    catchCudaError(hipEventRecord(device_start));

    betweennessCentralityKernel<<<1, MAX_THREAD_COUNT>>>(graph, device_bwCentrality, nodeCount, sigma, distance, dependency);
    hipDeviceSynchronize();

    // Timer
    catchCudaError(hipEventRecord(device_end));
    catchCudaError(hipEventSynchronize(device_end));
    hipEventElapsedTime(&device_time_taken, device_start, device_end);

    // Copy back and free memory
    catchCudaError(hipMemcpy(bwCentrality, device_bwCentrality, sizeof(double) * nodeCount, hipMemcpyDeviceToHost));
    catchCudaError(hipFree(device_bwCentrality));
    catchCudaError(hipFree(sigma));
    catchCudaError(hipFree(dependency));
    catchCudaError(hipFree(distance));
    return bwCentrality;
}

int main(int argc, char *argv[])
{

    if (argc < 2)
    {
        cout << "Please use correct format while execution" << endl;
        return 0;
    }

    // char choice;
    // cout << "Would you like to print the Graph Betweenness Centrality for all nodes? (y/n) ";
    // cin >> choice;

    freopen(argv[1], "r", stdin);

    Graph *host_graph = new Graph();
    Graph *device_graph;

    catchCudaError(hipMalloc((void **)&device_graph, sizeof(Graph)));
    host_graph->readGraph();

    int nodeCount = host_graph->getNodeCount();
    int edgeCount = host_graph->getEdgeCount();
    catchCudaError(hipMemcpy(device_graph, host_graph, sizeof(Graph), hipMemcpyHostToDevice));

    // Copy Adjancency List to device
    int *adjacencyList;
    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&adjacencyList, sizeof(int) * (2 * edgeCount + 1)));
    catchCudaError(hipMemcpy(adjacencyList, host_graph->adjacencyList, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));
    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->adjacencyList), &adjacencyList, sizeof(int *), hipMemcpyHostToDevice));

    // Copy Adjancency List Pointers to device
    int *adjacencyListPointers;
    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&adjacencyListPointers, sizeof(int) * (nodeCount + 1)));
    catchCudaError(hipMemcpy(adjacencyListPointers, host_graph->adjacencyListPointers, sizeof(int) * (nodeCount + 1), hipMemcpyHostToDevice));
    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->adjacencyListPointers), &adjacencyListPointers, sizeof(int *), hipMemcpyHostToDevice));

    double *bwCentrality = betweennessCentrality(device_graph, nodeCount);

    double maxBetweenness = -1;
    for (int i = 0; i < nodeCount; i++)
    {
        maxBetweenness = max(maxBetweenness, bwCentrality[i]);
        // if (choice == 'y' || choice == 'Y')
        //     printf("Node %d => Betweeness Centrality %0.2lf\n", i, bwCentrality[i]);
    }

    printf("Maximum Betweenness Centrality = %0.2lf\n", maxBetweenness);
    printf("Time Taken (Parallel) = %d ms\n", (int)device_time_taken);

    if (argc == 3)
    {
        freopen(argv[2], "w", stdout);
        for (int i = 0; i < nodeCount; i++)
            cout << bwCentrality[i] << " ";
        cout << endl;
    }

    // Free all memory
    catchCudaError(hipFree(adjacencyList));
    catchCudaError(hipFree(adjacencyListPointers));
    catchCudaError(hipFree(device_graph));
}